
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1e8;

  // Allocate host mem
  float *x = new float[N];
  float *y = new float[N];

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Prepare GPU mem
  float *dev_a, *dev_b;
  hipMalloc((void**)&dev_a, N*sizeof(float));
  hipMalloc((void**)&dev_b, N*sizeof(float));
  hipMemcpy(dev_a, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Run kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(N, dev_a, dev_b);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Get the results back
  hipMemcpy(y, dev_b, N*sizeof(float), hipMemcpyDeviceToHost);

  // Free GPU mem
  hipFree(dev_a);
  hipFree(dev_b);

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  delete [] x;
  delete [] y;

  return 0;
}
